#include "hip/hip_runtime.h"
#include "MemTimeTester.h"
#include "hip/hip_runtime.h"
#include ""
#include "obs_calc_fast.cuh"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/count.h>
#include <thrust/inner_product.h>
#include <thrust/functional.h>
#include <string>


#include <stdio.h>


int main()
{

	MemTimeTester timer;
	thrust::host_vector<int> state;
	std::ifstream file;
	std::string line;
	int Lx, Ly;
/*
	//check single site energy, correlation function
	std::cout << "Calculating Single Site energy and correlation\n\n";
	file.open("C:/Users/Matthew/Dropbox/Code/class_mc/windows/ClassMCTest/x64/Debug/fast_calc_testing/single_site/dump/state0.csv");
	if (file.is_open()) {
		file >> Lx >> Ly;
		state.resize(Lx * Ly);
		for (int i = 0; i < Lx; ++i) {
			for (int j = 0; j < Ly; ++j) {
				file >> state[i*Ly + j];
			}
		}
	}
	else {
		std::cout << "Error: input file not opened\n";
	}
	timer.flag_start_time("single site correlation measurement");
	thrust::host_vector<int>& state_ref = state;
	thrust::host_vector<double> corr(Lx*Ly);
	thrust::host_vector<double>& corr_ref = corr;
	calc_corr_fast_1site(corr_ref, state_ref, Ly);
	timer.flag_end_time("single site correlation measurement");
	std::cout << "\nSingle Site Correlation function:\n";
	for (int i = 0; i < corr.size(); ++i) {
		std::cout << corr[i] << ",";
	}
	std::cout << "\n";
	file.close();

	thrust::host_vector<double> interactions(Lx*Ly);
	thrust::host_vector<double>& int_ref = interactions;
	file.open("C:/Users/Matthew/Dropbox/Code/class_mc/windows/ClassMCTest/x64/Debug/fast_calc_testing/single_site/interactions.csv");
	if (file.is_open()) {
		for (int i = 0; i < Lx*Ly; ++i) {
			file >> interactions[i];
		}
	}
	else {
		std::cout << "Interactions file failed to open.\n";
	}
	file.close();
	std::cout << "\nInteractions:\n";
	for (int i = 0; i < interactions.size(); ++i) {
		std::cout << interactions[i] << ",";
	}
	std::cout << "\n";
	timer.flag_start_time("single site energy calculation");
	std::cout << "Energy calculated: " << calc_action_fast(corr_ref, int_ref) << "\n";
	timer.flag_end_time("single site energy calculation");

	std::cout << "Sx: " << 0.5*(1 - corr[1]) << "\n";

	//check double site energy, correlation function
	std::cout << "Calculating Double Site energy and correlation\n\n";
	file.open("C:/Users/Matthew/Dropbox/Code/class_mc/windows/ClassMCTest/x64/Debug/fast_calc_testing/double_site/dump/state0.csv");
	if (file.is_open()) {
		file >> Lx >> Ly;
		state.resize(Lx * Ly);
		for (int i = 0; i < Lx; ++i) {
			for (int j = 0; j < Ly; ++j) {
				file >> state[i*Ly + j];
			}
		}
	}
	else {
		std::cout << "Error: input file not opened\n";
	}
	file.close();
	timer.flag_start_time("double site correlation measurement");
	state_ref = state;
	corr.resize(Lx * Ly);
	corr_ref = corr;
	std::cout << "Lx, Ly: " << Lx << ", " << Ly << "\n";
	std::cout << "state size: " << state.size() << "\n";
	std::cout << "correlation size: " << corr.size() << "\n";
	calc_corr_fast_2site(corr_ref, state_ref, Ly);
	timer.flag_end_time("double site correlation measurement");
	std::cout << "Double Site Correlation function:\n";
	for (int i = 0; i < corr.size(); ++i) {
		std::cout << corr[i] << ",";
	}
	std::cout << "\n";
	file.close();

	file.open("C:/Users/Matthew/Dropbox/Code/class_mc/windows/ClassMCTest/x64/Debug/fast_calc_testing/double_site/interactions.csv");
	interactions.resize(Lx*Ly);
	int_ref = interactions;
	if (file.is_open()) {
		for (int i = 0; i < Lx; ++i) {
			for (int j = 0; j < Ly; ++j) {
				file >> interactions[i*Ly + j];
			}
			std::getline(file, line);
		}
	}
	else {
		std::cout << "Interactions file failed to open.\n";
	}
	file.close();
	std::cout << "\nInteractions:\n";
	for (int i = 0; i < interactions.size(); ++i) {
		std::cout << interactions[i] << ",";
	}
	std::cout << "\n";
	timer.flag_start_time("double site energy calculation");
	std::cout << "Energy calculated: " << calc_action_fast(corr_ref, int_ref) << "\n";
	timer.flag_end_time("double site energy calculation");

	std::cout << "Sx: " << 0.5*(1 - corr[1]) << "\n";

	//check average energy, correlation function
	std::cout << "Calculating average of 3 runs energy and correlation\n\n";
	int nruns = 3;
	char filename[300];
	Lx = 1, Ly = 100;
	state.resize(Lx * Ly);
	state_ref = state;
	corr.resize(Lx * Ly);
	thrust::device_vector<double> corr_temp = corr;
	thrust::host_vector<double>& corr_temp_ref = corr;
	thrust::device_vector<double> corr_total(Lx*Ly, 0.0);
	for (int run = 0; run < nruns; ++run) {
		sprintf(filename, "C:/Users/Matthew/Dropbox/Code/class_mc/windows/ClassMCTest/x64/Debug/fast_calc_testing/avg_states/dump/state%d.csv", run);
		file.open(filename);
		if (file.is_open()) {
			file >> Lx >> Ly;
			for (int i = 0; i < Lx; ++i) {
				for (int j = 0; j < Ly; ++j) {
					file >> state[i*Ly + j];
				}
			}
		}
		else {
			std::cout << "Error: input file not opened\n";
		}
		file.close();
		timer.flag_start_time("double site correlation measurement");
		calc_corr_fast_1site(corr_temp_ref, state_ref, Ly);
		corr_temp = corr_temp_ref;
		thrust::transform(corr_temp.begin(), corr_temp.end(), corr_total.begin(), corr_total.begin(), thrust::plus<double>());
		timer.flag_end_time("double site correlation measurement");
	}
	thrust::constant_iterator<double> factor(1.0 / 3.0);
	thrust::transform(corr_total.begin(), corr_total.end(), factor, corr_temp.begin(), thrust::multiplies<double>());
	corr = corr_temp;
	std::cout << "Double Site Correlation function:\n";
	for (int i = 0; i < corr.size(); ++i) {
		std::cout << corr[i] << ",";
	}
	std::cout << "\n";

	/*file.open("C:/Users/Matthew/Dropbox/Code/class_mc/windows/ClassMCTest/x64/Debug/fast_calc_testing/double_site/interactions.csv");
	interactions.resize(Lx*Ly);
	int_ref = interactions;
	if (file.is_open()) {
		for (int i = 0; i < Lx; ++i) {
			for (int j = 0; j < Ly; ++j) {
				file >> interactions[i*Ly + j];
			}
			std::getline(file, line);
		}
	}
	else {
		std::cout << "Interactions file failed to open.\n";
	}
	file.close();
	std::cout << "\nInteractions:\n";
	for (int i = 0; i < interactions.size(); ++i) {
		std::cout << interactions[i] << ",";
	}
	std::cout << "\n";
	timer.flag_start_time("double site energy calculation");
	std::cout << "Energy calculated: " << calc_action_fast(corr_ref, int_ref) << "\n";
	timer.flag_end_time("double site energy calculation");

	std::cout << "Sx: " << 0.5*(1 - corr[1]) << "\n";
*/
	//Perform the resampling procedure using the states saved in classmctest
	std::cout << "Creating resampling data\n";
	double base_alpha = 1.1, temp_mag;
	int num_alphas = 11;
	Ly = 100;
	int num_runs = 10000;
	char filename[300];
	std::vector<double> alphas(num_alphas);
	std::vector<double> avg_mag2s(num_alphas);
	std::vector<double> state_sums(num_runs);
	std::vector<double> mag2s(num_runs);
	thrust::host_vector<double> resample_interactions(Ly);
	thrust::host_vector<double> temp_corr(Ly);
	thrust::host_vector<double>& corr_ref = temp_corr;
	thrust::host_vector<double>& int_ref = resample_interactions;
	thrust::host_vector<int>& state_ref = state;
	file.open("C:/Users/Matthew/Dropbox/Code/class_mc/windows/ClassMCTest/x64/Debug/fast_calc_testing/resampling/alphas.txt");
	if (file.is_open()) {
		for (int i = 0; i < num_alphas; ++i) {
			file >> alphas[i];
		}
	}
	else {
		std::cout << "Error: alpha file not opened\n";
	}
	file.close();

	file.open("C:/Users/Matthew/Dropbox/Code/class_mc/windows/ClassMCTest/x64/Debug/fast_calc_testing/resampling/resample_interactions.csv");
	if (file.is_open()) {
		for (int i = 0; i < Ly; ++i) {
			file >> resample_interactions[i];
		}
	}
	else {
		std::cout << "Error: interactions file not opened\n";
	}
	file.close();

	for (int run = 0; run < num_runs; ++run) {
		sprintf(filename, "C:/Users/Matthew/Dropbox/Code/class_mc/windows/ClassMCTest/x64/Debug/fast_calc_testing/resampling/state%d.csv", run);
		file.open(filename);
		if (file.is_open()) {
			file >> Lx >> Ly;
			state.resize(Lx * Ly);
			for (int i = 0; i < Lx; ++i) {
				for (int j = 0; j < Ly; ++j) {
					file >> state[i*Ly + j];
				}
			}
		}
		else {
			std::cout << "Error: state file not opened\n";
		}

		calc_corr_fast_1site(corr_ref, state_ref, Ly);
		state_sums[run] = calc_action_fast(corr_ref, int_ref);
		temp_mag = thrust::inner_product(state.begin(), state.end(), thrust::make_constant_iterator(1.0), 0.0);
		mag2s[run] = temp_mag*temp_mag/Ly/Ly;
		file.close();
	}
	//redo averages with different parameters
	double m2avg = 0;
	for (int i = 0; i < mag2s.size(); ++i) {
		m2avg += mag2s[i];
	}
	std::cout << "avg mag2 for alpha = 1: " << m2avg/mag2s.size() << "\n";

	//find the initial histogram - assume relatively gaussian
	//use the max, min, and mean
	double max = state_sums[0], min = state_sums[0], sum_temp = 0;
	for (int i = 0; i < state_sums.size(); ++i) {
		sum_temp = state_sums[i];
		if (max < sum_temp) { max = sum_temp; }
		if (min > sum_temp) { min = sum_temp; }
	}
	int num_bins = state_sums.size() / 10;
	double bin_step = (max - min) / (num_bins - 1);
	std::vector<int> histogram(num_bins);
	int bin;
	for (int i = 0; i < state_sums.size(); ++i) {
		bin = (int)((state_sums[i] - min) / bin_step);
		if (bin >= 0 && bin < num_bins) {
			histogram[bin] += 1;
		}
		else {
			std::cout << "Error: bin index out of bounds\n";
		}
	}
	for (int i = 0; i < num_bins; ++i) {
		std::cout << histogram[i] << ",";
	}
	std::cout << "\n";

	//resample using other alphas
	double part_func, temp_prob;
	for (int i = 0; i < alphas.size(); ++i) {
		temp_mag = 0.0;
		part_func = 0.0;
		//find the approximate partition function for the new value of alpha
		for (int s = 0; s < num_runs; ++s) {
			bin = (int)((state_sums[s] - min) / bin_step);
			temp_prob = (1.0*histogram[bin]) / num_runs;
			part_func += temp_prob*exp(-(alphas[i] - base_alpha) * state_sums[s]);
		}
		//find the average mag^2 with the new probability distribution
		for (int s = 0; s < num_runs; ++s) {
			bin = (int)((state_sums[s] - min) / bin_step);
			temp_prob = (1.0*histogram[bin]) / num_runs;
			temp_mag += mag2s[s] * temp_prob * exp(-(alphas[i] - base_alpha) * state_sums[s]) / part_func;
		}
		std::cout << "partition function: " << part_func << " and mag2: " << temp_mag << "\n";
		avg_mag2s[i] = temp_mag;
	}
	std::ofstream outfile;
	outfile.open("C:/Users/Matthew/Dropbox/Code/class_mc/windows/ClassMCTest/x64/Debug/fast_calc_testing/resampling/resample_results.csv");
	for (int i = 0; i < alphas.size(); ++i) {
		outfile << alphas[i] << ",";
	}
	outfile << "\n";
	for (int i = 0; i < alphas.size(); ++i) {
		outfile << avg_mag2s[i] << ",";
	}
	outfile.close();

	timer.print_timers();
    return 0;
}
