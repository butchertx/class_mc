#include "hip/hip_runtime.h"
#include "obs_calc_fast.cuh"
#include <sstream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/inner_product.h>
#include <thrust/copy.h>
#include <thrust/reduce.h>
#include <thrust/gather.h>
#include <thrust/scan.h>
#include <thrust/fill.h>
#include <thrust/copy.h>

#include <iterator>
#include <iostream>

template <typename InputIterator1,
	typename InputIterator2,
	typename OutputIterator>
	OutputIterator expand(InputIterator1 first1,
		InputIterator1 last1,
		InputIterator2 first2,
		OutputIterator output)
{
	typedef typename thrust::iterator_difference<InputIterator1>::type difference_type;

	difference_type input_size = thrust::distance(first1, last1);
	difference_type output_size = thrust::reduce(first1, last1);

	// scan the counts to obtain output offsets for each input element
	thrust::device_vector<difference_type> output_offsets(input_size, 0);
	thrust::exclusive_scan(first1, last1, output_offsets.begin());

	// scatter the nonzero counts into their corresponding output positions
	thrust::device_vector<difference_type> output_indices(output_size, 0);
	thrust::scatter_if
	(thrust::counting_iterator<difference_type>(0),
		thrust::counting_iterator<difference_type>(input_size),
		output_offsets.begin(),
		first1,
		output_indices.begin());

	// compute max-scan over the output indices, filling in the holes
	thrust::inclusive_scan
	(output_indices.begin(),
		output_indices.end(),
		output_indices.begin(),
		thrust::maximum<difference_type>());

	// gather input values according to index array (output = first2[output_indices])
	OutputIterator output_end = output; thrust::advance(output_end, output_size);
	thrust::gather(output_indices.begin(),
		output_indices.end(),
		first2,
		output);

	// return output + output_size
	thrust::advance(output, output_size);
	return output;
}


std::vector<int> calc_corr_fast(std::ifstream* file_p) {
	int Lx, Ly;
	thrust::host_vector<int> state;

	if (file_p->is_open()) {
		*file_p >> Lx >> Ly;
		state.resize(Lx * Ly);
		for (int i = 0; i < Lx; ++i) {
			for (int j = 0; j < Ly; ++j) {
				*file_p >> state[i*Ly + j];
			}
		}
	}
	else {
		std::cout << "Error: input file not opened\n";
	}
	////output state vector to check
	//for (int i = 0; i < Lx; ++i) {
	//	for (int j = 0; j < Ly; ++j) {
	//		std::cout << state[i*Ly + j] << ", ";
	//	}
	//	std::cout << "\n";
	//}

	//begin calculation
	//copy to device
	thrust::device_vector<int> d_state = state;
	thrust::device_vector<int> corr(Lx*Ly, 0);


	////output state vector to check
	//for (int i = 0; i < Lx; ++i) {
	//	for (int j = 0; j < Ly; ++j) {
	//		std::cout << d_state[i*Ly + j] << ", ";
	//	}
	//	std::cout << "\n";
	//}

	//calculate correlation
	//only works for Lx = 1 right now
	for (int i = 0; i < Ly; ++i) {
		corr[i] = thrust::inner_product(d_state.begin(), d_state.end() - i, d_state.begin() + i, 0) + thrust::inner_product(d_state.begin(), d_state.begin() + i, d_state.end() - i, 0);
	}

	//for (int i = 0; i < Lx; ++i) {
	//	for (int j = 0; j < Ly; ++j) {
	//		std::cout << corr[i*Ly + j] << ", ";
	//	}
	//	std::cout << "\n";
	//}

	return{ 0 };
}