#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <thrust/host_vector.h>
#include "MemTimeTester.h"
#include "obs_calc_fast.cuh"
extern "C" {
#include "random.h"
}
/**
For my GPU, the maximum threads per multiprocessor is 2048, max per block is 1024, and the max dimensions are (1024, 1024, 64)
Maximum shared memory per block is 49152 bytes, L2 Cache is 2097152 bytes, 15 multiprocessors, 128 CUDA cores/MP
Cuda capability: 6.1, CUDA driver version 8.0
**/

double calc_action_slow(thrust::host_vector<double>& lat, thrust::host_vector<double>& interactions, int Lx, int Ly) {
	//essentially, the time-averaged energy of a given set of quantum fluctuations.  Calculate the action then divide by beta
	double S = 0;
	double s1;
	for (int i = 0; i < Lx; ++i) {
		for (int j = 0; j < Ly; ++j) {
			s1 = lat[i*Ly + j];
			for (int m = 0; m < Lx; ++m) {
				for (int n = 0; n < Ly; ++n) {
					S += s1 * lat[m*Ly + n] * interactions[((m - i + Lx) % Lx)*Ly + ((n - j + Ly) % Ly)];
				}
			}
		}
	}

	return 0.5*S;
}

thrust::host_vector<double> rand_vector(int length) {
	thrust::host_vector<double> result(length);
	for (int i = 0; i < length; ++i) {
		result[i] = drand1_();
	}
	return result;
}

thrust::host_vector<double> transpose(thrust::host_vector<double> s, int Lx, int Ly) {
	//take a vector indexed like s(x, y) = s[x*Ly + y] and make it s(x,y) = s[y*Lx + x]
	thrust::host_vector<double> new_s(Lx*Ly);
	for (int x = 0; x < Lx; ++x) {
		for (int y = 0; y < Ly; ++y) {
			new_s[y*Lx + x] = s[x*Ly + y];
		}
	}
	return new_s;
}

int main() {
	std::cout << "Timing different partitions of fast calc kernel\n";
	MemTimeTester timer;
	int seed = 1892347;
	rand_init_(&seed);
	double fast_action;
	double cufft_action;
	double slow_action;


	//create different sizes of lattice to calculate, then populate with random numbers
	//sizes to test
	//2x128, 2x1024, 2x16384
	//32x32, 32x64, 32x1024
	//1024x1024

	thrust::host_vector<double> vec2x128 = rand_vector(256),
		int2x128 = rand_vector(256),
		vec32x32 = rand_vector(32 * 32),
		int32x32 = rand_vector(32 * 32),
		vec32x64 = rand_vector(32 * 64),
		int32x64 = rand_vector(32 * 64),
		vec32x1024 = rand_vector(32 * 1024),
		int32x1024 = rand_vector(32 * 1024);

	//2x128
	dim3 threads(128, 1, 1);
	thrust::host_vector<double>& sref = vec2x128;
	thrust::host_vector<double>& iref = int2x128;
	timer.flag_start_time("2x128 {2,0,0} {128,0,0}");
	fast_action = thrust_calc_action_general(sref, iref, 2, 128, threads);
	timer.flag_end_time("2x128 {2,0,0} {128,0,0}");
	slow_action = calc_action_slow(sref, iref, 2, 128);
	cufft_action = cufft_calc_action(sref, iref, 2, 128) / 256.0;
	std::cout << "Fast action = " << fast_action << ", slow action = " << slow_action << ", cufft action: " << cufft_action << "\n";
	show_memory();

	//32x32
	dim3 threads2(4, 4, 2);
	sref = vec32x32;
	iref = int32x32;
	timer.flag_start_time("32x32 {32,0,0} {32,0,0}");
	fast_action = thrust_calc_action_general(sref, iref, 32, 32, threads2);
	timer.flag_end_time("32x32 {32,0,0} {32,0,0}");
	slow_action = calc_action_slow(sref, iref, 32, 32);
	cufft_action = cufft_calc_action(sref, iref, 32, 32) / (32.0*32);
	std::cout << "Fast action = " << fast_action << ", slow action = " << slow_action << ", cufft action: " << cufft_action << "\n";
	show_memory();


	//32x64
	dim3 threads4(4, 4, 4);
	sref = vec32x64;
	iref = int32x64;
	timer.flag_start_time("32x64 {32,0,0} {32,0,0}");
	fast_action = thrust_calc_action_general(sref, iref, 32, 64, threads4);
	timer.flag_end_time("32x64 {32,0,0} {32,0,0}");
	slow_action = calc_action_slow(sref, iref, 32, 64);
	cufft_action = cufft_calc_action(sref, iref, 32, 64) / (32.0 * 64);
	std::cout << "Fast action = " << fast_action << ", slow action = " << slow_action << ", cufft action: " << cufft_action << "\n";
	show_memory();

	//32x1024
	dim3 threads3(32, 32, 1);
	sref = vec32x1024;
	iref = int32x1024;
	timer.flag_start_time("big");
	fast_action = thrust_calc_action_general(sref, iref, 32, 1024, threads3);
	timer.flag_end_time("big");
	timer.flag_start_time("CUFFT");
	cufft_action = cufft_calc_action(sref, iref, 32, 1024)/(32.0*1024);
	timer.flag_end_time("CUFFT");
	std::cout << "Fast action = " << fast_action << ", cufft action: " << cufft_action << "\n";
	show_memory();

	timer.print_timers();

	return 0;
}